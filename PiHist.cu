#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>

#define BLOCK_SIZE 256

__global__ void hist_kernel(const char *digits, int len, int *counts);
int main(int argc, char **argv)
{
    if (argc != 3)
    {
        fprintf(stderr, "Usage: %s <filename> <num_digits>\n", argv[0]);
        exit(1);
    }

    char *filename = argv[1];
    int num_digits = atoi(argv[2]);
    if (num_digits <= 0)
    {
        fprintf(stderr, "Number of digits must be positive\n");
        exit(1);
    }

    FILE *fp = fopen(filename, "r");
    if (!fp)
    {
        fprintf(stderr, "Failed to open file %s\n", filename);
        exit(1);
    }

    char *digits = (char *)malloc(num_digits * sizeof(char));
    if (!digits)
    {
        fprintf(stderr, "Failed to allocate memory\n");
        exit(1);
    }
    int len = fread(digits, sizeof(char), num_digits, fp);
    if (len != num_digits)
    {
        fprintf(stderr, "Failed to read expected number of digits\n");
        exit(1);
    }

    int *counts = (int *)calloc(10, sizeof(int));
    if (!counts)
    {
        fprintf(stderr, "Failed to allocate memory\n");
        exit(1);
    }

    char *d_digits;
    int *d_counts;
    hipMalloc((void **)&d_digits, num_digits * sizeof(char));
    hipMalloc((void **)&d_counts, 10 * sizeof(int));
    hipMemcpy(d_digits, digits, num_digits * sizeof(char), hipMemcpyHostToDevice);
    hipMemcpy(d_counts, counts, 10 * sizeof(int), hipMemcpyHostToDevice);

    int num_blocks = (num_digits + BLOCK_SIZE - 1) / BLOCK_SIZE;
    hist_kernel<<<num_blocks, BLOCK_SIZE>>>(d_digits, num_digits, d_counts);

    hipMemcpy(counts, d_counts, 10 * sizeof(int), hipMemcpyDeviceToHost);
    printf("Digit counts: ");
    for (int i = 0; i < 10; i++)
    {
        printf("%d:%d ", i, counts[i]);
    }
    printf("\n");

    FILE *output_file = fopen("digit_counts.csv", "a");
    if (!output_file)
    {
        fprintf(stderr, "Failed to create output file\n");
        exit(1);
    }
    fprintf(output_file, "Number of digits: %d\n", num_digits);
    fprintf(output_file, "digit,frequency\n");
    for (int i = 0; i < 10; i++)
    {
        fprintf(output_file, "%d,%d\n", i, counts[i]);
    }
    fprintf(output_file, "\n");
    fclose(output_file);

    hipFree(d_digits);
    hipFree(d_counts);
    free(digits);
    free(counts);
    fclose(fp);

    return 0;
}
__global__ void hist_kernel(const char *digits, int len, int *counts)
{
    __shared__ int block_counts[10];
    for (int i = threadIdx.x; i < 10; i += blockDim.x)
    {
        block_counts[i] = 0;
    }
    __syncthreads();

    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    while (idx < len)
    {
        int digit = digits[idx] - '0';
        atomicAdd(&block_counts[digit], 1);
        idx += gridDim.x * blockDim.x;
    }
    __syncthreads();

    for (int i = threadIdx.x; i < 10; i += blockDim.x)
    {
        atomicAdd(&counts[i], block_counts[i]);
    }
}